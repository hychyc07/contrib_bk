#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

// includes, kernels
#include <cppIntegration_kernel.cu>
//
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void
computeGold(char* reference, char* idata, const unsigned int len);
extern "C" void
computeGold2(int2* reference, int2* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" void
runTest(const int argc, const char** argv, char* data, int2* data_int2, unsigned int len)
{

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, (char**)argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    const unsigned int num_threads = len / 4;
    cutilCondition(0 == (len % 4));
    const unsigned int mem_size = sizeof(char) * len;
    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    char* d_data;
    cutilSafeCall(hipMalloc((void**) &d_data, mem_size));
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_data, data, mem_size,
                            hipMemcpyHostToDevice) );
    // allocate device memory for int2 version
    int2* d_data_int2;
    cutilSafeCall(hipMalloc((void**) &d_data_int2, mem_size_int2));
    // copy host memory to device
    cutilSafeCall(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
                            hipMemcpyHostToDevice) );

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
    // execute the kernel
    kernel<<< grid, threads >>>((int*) d_data);
    kernel2<<< grid, threads2 >>>(d_data_int2);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // compute reference solutions
    char* reference = (char*) malloc(mem_size);
    computeGold(reference, data, len);
    int2* reference2 = (int2*) malloc(mem_size_int2);
    computeGold2(reference2, data_int2, len);

    // copy results from device to host
    cutilSafeCall(hipMemcpy(data, d_data, mem_size,
                            hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
                            hipMemcpyDeviceToHost));

    // check result
    bool success = true;
    for(unsigned int i = 0; i < len; i++ )
    {
        if( reference[i] != data[i] || 
	    reference2[i].x != data_int2[i].x || 
	    reference2[i].y != data_int2[i].y)
            success = false;
    }
    printf("%s\n", success ? "PASSED" : "FAILED");

    // cleanup memory
    cutilSafeCall(hipFree(d_data));
    cutilSafeCall(hipFree(d_data_int2));
    free(reference);
    free(reference2);

    hipDeviceReset();
}
